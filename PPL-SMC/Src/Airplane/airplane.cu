#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <random>
#include <time.h>

#include "../benchmark_suite.hpp"

#include "../Smc/smc.cuh"
#include "../Smc/smcImpl.cuh"
#include "airplane.cuh"
#include "airplaneUtils.cuh"
#include "../Utils/distributions.cuh"
#include "../Utils/array.cuh"
#include "../Utils/misc.cuh"

// nvcc -arch=sm_61 -rdc=true Src/Airplane/*.cu Src/Utils/*.cpp -o smc.exe -lcudadevrt -std=c++11 -O3 -D GPU

using namespace std;


floating_t planeX[TIME_STEPS];

BBLOCK_DATA(planeObs, floating_t, TIME_STEPS)
BBLOCK_DATA(mapApprox, floating_t, MAP_SIZE)


void initAirplane() {

    initMap(mapApprox);

    initObservations(planeX, planeObs, mapApprox);

    // Copy data to device pointers, so that they can be accessed from kernels
    COPY_DATA_GPU(planeObs, floating_t, TIME_STEPS)
    COPY_DATA_GPU(mapApprox, floating_t, MAP_SIZE)
}


BBLOCK(particleInit, progState_t, {

    PSTATE.x = sampleUniform(particles, i, 0, MAP_SIZE);

    PC = 1;
    RESAMPLE = false;
})

BBLOCK(propagateAndWeight, progState_t, {

    // Propagate
    PSTATE.x += sampleNormal(particles, i, VELOCITY, TRANSITION_STD);

    // Weight
    WEIGHT(logNormalPDFObs(DATA_POINTER(planeObs)[t], mapLookupApprox(DATA_POINTER(mapApprox), PSTATE.x)));

    if(t >= TIME_STEPS - 1)
        PC = 2;

    RESAMPLE = true;
})

STATUSFUNC({
    // Checks how many particles are close to actual airplane to check for correctness
// JW: This has been commented out such that time wasted here is not included
//     in benchmarking results.
//    int numParticlesClose = 0;
//    floating_t minX = 999999;
//    floating_t maxX = -1;
//    for (int i = 0; i < NUM_PARTICLES; i++) {
//        floating_t particleX = PSTATE.x;
//        if(abs(particleX - planeX[t]) < 10)
//            numParticlesClose++;
//        minX = min(minX, particleX);
//        maxX = max(maxX, particleX);
//    }
//
//    cout << "TimeStep " << t << ", Num particles close to target: " << 100 * static_cast<floating_t>(numParticlesClose) / NUM_PARTICLES << "%, MinX: " << minX << ", MaxX: " << maxX << endl;
})

void bm_prepare(void);
void bm_run(void);
void bm_cleanup(void);

int main(int argc, char** argv) {

    initAirplane();

    BENCHMARK(bm_prepare, bm_run, bm_cleanup);
}

// Benchmarking wrappers

void bm_prepare(void)
{
    // do nothing
}

void bm_run(void)
{
    SMCSTART(progState_t)

    INITBBLOCK(particleInit, progState_t)
    INITBBLOCK(propagateAndWeight, progState_t)

    SMCEND(progState_t)
}

void bm_cleanup(void)
{
    // do nothing
}
